#include "hip/hip_runtime.h"
#include "serpent.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#ifndef SUBKEY_LENGTH
#define SUBKEY_LENGTH 132
#endif


// order of output from S-box functions
#define beforeS0(f) f(0,a,b,c,d,e)
#define afterS0(f) f(1,b,e,c,a,d)
#define afterS1(f) f(2,c,b,a,e,d)
#define afterS2(f) f(3,a,e,b,d,c)
#define afterS3(f) f(4,e,b,d,c,a)
#define afterS4(f) f(5,b,a,e,c,d)
#define afterS5(f) f(6,a,c,b,e,d)
#define afterS6(f) f(7,a,c,d,b,e)
#define afterS7(f) f(8,d,e,b,a,c)

// order of output from inverse S-box functions
#define beforeI7(f) f(8,a,b,c,d,e)
#define afterI7(f) f(7,d,a,b,e,c)
#define afterI6(f) f(6,a,b,c,e,d)
#define afterI5(f) f(5,b,d,e,c,a)
#define afterI4(f) f(4,b,c,e,a,d)
#define afterI3(f) f(3,a,b,e,c,d)
#define afterI2(f) f(2,b,d,e,c,a)
#define afterI1(f) f(1,a,b,c,e,d)
#define afterI0(f) f(0,a,d,b,e,c)

// The linear transformation.
#define linear_transformation(i,a,b,c,d,e) {\
        a = rotl_fixed(a, 13);   \
        c = rotl_fixed(c, 3);    \
        d = rotl_fixed(d ^ c ^ (a << 3), 7);     \
        b = rotl_fixed(b ^ a ^ c, 1);    \
        a = rotl_fixed(a ^ b ^ d, 5);       \
        c = rotl_fixed(c ^ d ^ (b << 7), 22);}

// The inverse linear transformation.
#define inverse_linear_transformation(i,a,b,c,d,e)        {\
        c = rotr_fixed(c, 22);   \
        a = rotr_fixed(a, 5);    \
        c ^= d ^ (b << 7);      \
        a ^= b ^ d;             \
        b = rotr_fixed(b, 1);    \
        d = rotr_fixed(d, 7) ^ c ^ (a << 3);     \
        b ^= a ^ c;             \
        c = rotr_fixed(c, 3);    \
        a = rotr_fixed(a, 13);}

#define S0(i, r0, r1, r2, r3, r4) \
       {           \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r4 ^= r2;   \
    r1 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r4;   \
    r4 ^= r3;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 ^= r4;   \
    r4 = ~r4;      \
    r4 |= r1;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r3 |= r0;   \
    r1 ^= r3;   \
    r4 ^= r3;   \
            }

#define I0(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r1;   \
    r1 |= r0;   \
    r4 = ~r4;      \
    r1 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r4 ^= r0;   \
    r0 |= r1;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r2 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r1;   \
    r2 &= r3;   \
    r4 ^= r2;   \
            }

#define S1(i, r0, r1, r2, r3, r4) \
       {           \
    r0 = ~r0;      \
    r2 = ~r2;      \
    r4 = r0;   \
    r0 &= r1;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r3 ^= r2;   \
    r1 ^= r0;   \
    r0 ^= r4;   \
    r4 |= r1;   \
    r1 ^= r3;   \
    r2 |= r0;   \
    r2 &= r4;   \
    r0 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r2;   \
    r0 ^= r4;   \
            }

#define I1(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r1;   \
    r1 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r3 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r3;   \
    r0 ^= r4;   \
    r0 |= r2;   \
    r1 ^= r3;   \
    r0 ^= r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
    r4 = ~r4;      \
    r4 ^= r1;   \
    r1 |= r0;   \
    r1 ^= r0;   \
    r1 |= r4;   \
    r3 ^= r1;   \
            }

#define S2(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 &= r2;   \
    r0 ^= r3;   \
    r2 ^= r1;   \
    r2 ^= r0;   \
    r3 |= r4;   \
    r3 ^= r1;   \
    r4 ^= r2;   \
    r1 = r3;   \
    r3 |= r4;   \
    r3 ^= r0;   \
    r0 &= r1;   \
    r4 ^= r0;   \
    r1 ^= r3;   \
    r1 ^= r4;   \
    r4 = ~r4;      \
            }

#define I2(i, r0, r1, r2, r3, r4) \
       {           \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r3;   \
    r3 &= r2;   \
    r3 ^= r1;   \
    r1 |= r2;   \
    r1 ^= r4;   \
    r4 &= r3;   \
    r2 ^= r3;   \
    r4 &= r0;   \
    r4 ^= r2;   \
    r2 &= r1;   \
    r2 |= r0;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r0 ^= r3;   \
    r0 &= r1;   \
    r3 ^= r4;   \
    r3 ^= r0;   \
            }

#define S3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r0;   \
    r0 |= r3;   \
    r3 ^= r1;   \
    r1 &= r4;   \
    r4 ^= r2;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r4 |= r1;   \
    r3 ^= r4;   \
    r0 ^= r1;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r1 |= r0;   \
    r1 ^= r2;   \
    r0 ^= r3;   \
    r2 = r1;   \
    r1 |= r3;   \
    r1 ^= r0;   \
            }

#define I3(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r0;   \
    r0 &= r4;   \
    r4 ^= r3;   \
    r3 |= r1;   \
    r3 ^= r2;   \
    r0 ^= r4;   \
    r2 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r1;   \
    r4 ^= r2;   \
    r2 &= r3;   \
    r1 |= r3;   \
    r1 ^= r2;   \
    r4 ^= r0;   \
    r2 ^= r4;   \
            }

#define S4(i, r0, r1, r2, r3, r4) \
       {           \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r2 ^= r3;   \
    r3 ^= r0;   \
    r4 = r1;   \
    r1 &= r3;   \
    r1 ^= r2;   \
    r4 ^= r3;   \
    r0 ^= r4;   \
    r2 &= r4;   \
    r2 ^= r0;   \
    r0 &= r1;   \
    r3 ^= r0;   \
    r4 |= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r2 &= r3;   \
    r0 = ~r0;      \
    r4 ^= r2;   \
            }

#define I4(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r3;   \
    r2 ^= r1;   \
    r1 |= r3;   \
    r1 &= r0;   \
    r4 ^= r2;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r0 = ~r0;      \
    r3 ^= r4;   \
    r1 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r2;   \
    r0 ^= r1;   \
    r2 &= r0;   \
    r3 ^= r0;   \
    r2 ^= r4;   \
    r2 |= r3;   \
    r3 ^= r0;   \
    r2 ^= r1;   \
            }

#define S5(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r1;   \
    r1 ^= r3;   \
    r3 = ~r3;      \
    r4 = r1;   \
    r1 &= r0;   \
    r2 ^= r3;   \
    r1 ^= r2;   \
    r2 |= r4;   \
    r4 ^= r3;   \
    r3 &= r1;   \
    r3 ^= r0;   \
    r4 ^= r1;   \
    r4 ^= r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r0 ^= r4;   \
    r4 |= r3;   \
    r2 ^= r4;   \
            }

#define I5(i, r0, r1, r2, r3, r4) \
       {           \
    r1 = ~r1;      \
    r4 = r3;   \
    r2 ^= r1;   \
    r3 |= r0;   \
    r3 ^= r2;   \
    r2 |= r1;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 ^= r4;   \
    r4 |= r0;   \
    r4 ^= r1;   \
    r1 &= r2;   \
    r1 ^= r3;   \
    r4 ^= r2;   \
    r3 &= r4;   \
    r4 ^= r1;   \
    r3 ^= r0;   \
    r3 ^= r4;   \
    r4 = ~r4;      \
            }

#define S6(i, r0, r1, r2, r3, r4) \
       {           \
    r2 = ~r2;      \
    r4 = r3;   \
    r3 &= r0;   \
    r0 ^= r4;   \
    r3 ^= r2;   \
    r2 |= r4;   \
    r1 ^= r3;   \
    r2 ^= r0;   \
    r0 |= r1;   \
    r2 ^= r1;   \
    r4 ^= r0;   \
    r0 |= r3;   \
    r0 ^= r2;   \
    r4 ^= r3;   \
    r4 ^= r0;   \
    r3 = ~r3;      \
    r2 &= r4;   \
    r2 ^= r3;   \
            }

#define I6(i, r0, r1, r2, r3, r4) \
       {           \
    r0 ^= r2;   \
    r4 = r2;   \
    r2 &= r0;   \
    r4 ^= r3;   \
    r2 = ~r2;      \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r4 |= r0;   \
    r0 ^= r2;   \
    r3 ^= r4;   \
    r4 ^= r1;   \
    r1 &= r3;   \
    r1 ^= r0;   \
    r0 ^= r3;   \
    r0 |= r2;   \
    r3 ^= r1;   \
    r4 ^= r0;   \
            }

#define S7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 &= r1;   \
    r2 ^= r3;   \
    r3 &= r1;   \
    r4 ^= r2;   \
    r2 ^= r1;   \
    r1 ^= r0;   \
    r0 |= r4;   \
    r0 ^= r2;   \
    r3 ^= r1;   \
    r2 ^= r3;   \
    r3 &= r0;   \
    r3 ^= r4;   \
    r4 ^= r2;   \
    r2 &= r0;   \
    r4 = ~r4;      \
    r2 ^= r4;   \
    r4 &= r0;   \
    r1 ^= r3;   \
    r4 ^= r1;   \
            }

#define I7(i, r0, r1, r2, r3, r4) \
       {           \
    r4 = r2;   \
    r2 ^= r0;   \
    r0 &= r3;   \
    r2 = ~r2;      \
    r4 |= r3;   \
    r3 ^= r1;   \
    r1 |= r0;   \
    r0 ^= r2;   \
    r2 &= r4;   \
    r1 ^= r2;   \
    r2 ^= r0;   \
    r0 |= r2;   \
    r3 &= r4;   \
    r0 ^= r3;   \
    r4 ^= r1;   \
    r3 ^= r4;   \
    r4 |= r0;   \
    r3 ^= r2;   \
    r4 ^= r2;   \
            }

// key xor
#define KX(r, a, b, c, d, e)    {\
        a ^= subkey[4 * r + 0]; \
        b ^= subkey[4 * r + 1]; \
        c ^= subkey[4 * r + 2]; \
        d ^= subkey[4 * r + 3];}

/**	Decrypt a single block on the device.
 */
__device__ void serpent_cuda_decrypt_block(block128* block, uint32* subkey);


/**	Decrypt the specified array of blocks with the specified subkey through a CUDA thread.
 */
__global__ void serpent_cuda_decrypt_blocks(block128* cuda_blocks, uint32* subkey, int block_count, int blocks_per_thread );


/**	Encrypt a single block on the device.
 */
__device__ void serpent_cuda_encrypt_block(block128* block, uint32* subkey);


/**	Encrypt the specified array of blocks with the specified subkey through a CUDA thread.
 */
__global__ void serpent_cuda_encrypt_blocks(block128* cuda_blocks, uint32* subkey, int block_count, int blocks_per_thread );


/**	Flip the bytes of the specified 32-bit unsigned integer.
 *	@return	A 32-bit unsigned integer with the bytes mirrored.
 */
__device__ uint32 mirror_bytes32_cu(uint32 x);


// Constant variables must be declared with a static scope...
__device__ __constant__ uint32 cuda_subkey[SUBKEY_LENGTH];


__device__ void serpent_cuda_decrypt_block(block128* block, uint32* subkey) {
	uint32 a, b, c, d, e;
	int j;

	// Change to little endian.
        a = mirror_bytes32_cu(block->x0);
        b = mirror_bytes32_cu(block->x1);
        c = mirror_bytes32_cu(block->x2);
        d = mirror_bytes32_cu(block->x3);

	// Decrypt the current block.
	j = 4;
	subkey += 96;
	beforeI7(KX);
	goto start;
	do
	{
		c = b;
		b = d;
		d = e;
		subkey -= 32;
		beforeI7(inverse_linear_transformation);
	start:
		beforeI7(I7); afterI7(KX);
		afterI7(inverse_linear_transformation); afterI7(I6); afterI6(KX);
		afterI6(inverse_linear_transformation); afterI6(I5); afterI5(KX);
		afterI5(inverse_linear_transformation); afterI5(I4); afterI4(KX);
		afterI4(inverse_linear_transformation); afterI4(I3); afterI3(KX);
		afterI3(inverse_linear_transformation); afterI3(I2); afterI2(KX);
		afterI2(inverse_linear_transformation); afterI2(I1); afterI1(KX);
		afterI1(inverse_linear_transformation); afterI1(I0); afterI0(KX);
	}
	while (--j != 0);

	// Restore to big endian based on algorithm-defined order.
	block->x0 = mirror_bytes32_cu(a);
	block->x1 = mirror_bytes32_cu(d);
	block->x2 = mirror_bytes32_cu(b);
	block->x3 = mirror_bytes32_cu(e);
}


__global__ void serpent_cuda_decrypt_blocks( block128* cuda_blocks, int block_count, int blocks_per_thread ) {
	int index = (blockIdx.x * blockDim.x * blocks_per_thread) + (threadIdx.x * blocks_per_thread); // (beginning of multiprocessor segment) + (segment index).
	int i;

	// Encrypted the minimal number of blocks.
	for ( i = 0; i < blocks_per_thread; i++ ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index + i]), cuda_subkey);
	}

	// Encrypt the extra blocks that fall outside the minimal number of block.s
	index = ( gridDim.x * blockDim.x * blocks_per_thread ) + ((blockIdx.x * blockDim.x) + threadIdx.x); // (end of array) + (absolute thread #).
	if ( index < block_count ) {
		serpent_cuda_decrypt_block(&(cuda_blocks[index]), cuda_subkey);
	}
}


__device__ void serpent_cuda_encrypt_block(block128* block, uint32* subkey) {
	uint32 a, b, c, d, e;
	int j;

	// Change to little endian.
	a = mirror_bytes32_cu(block->x0);
	b = mirror_bytes32_cu(block->x1);
	c = mirror_bytes32_cu(block->x2);
	d = mirror_bytes32_cu(block->x3);

	// Encrypt the current block.
	j = 1;
	do {
		beforeS0(KX); beforeS0(S0); afterS0(linear_transformation);
		afterS0(KX); afterS0(S1); afterS1(linear_transformation);
		afterS1(KX); afterS1(S2); afterS2(linear_transformation);
		afterS2(KX); afterS2(S3); afterS3(linear_transformation);
		afterS3(KX); afterS3(S4); afterS4(linear_transformation);
		afterS4(KX); afterS4(S5); afterS5(linear_transformation);
		afterS5(KX); afterS5(S6); afterS6(linear_transformation);
		afterS6(KX); afterS6(S7);

		if (j == 4)
			break;

		++j;
		c = b;
		b = e;
		e = d;
		d = a;
		a = e;
		subkey += 32;
		beforeS0(linear_transformation);
	} while (1);
	afterS7(KX);

	// Restore to big endian based on algorithm-defined order.
	block->x0 = mirror_bytes32_cu(d);
	block->x1 = mirror_bytes32_cu(e);
	block->x2 = mirror_bytes32_cu(b);
	block->x3 = mirror_bytes32_cu(a);
}


__global__ void serpent_cuda_encrypt_blocks( block128* cuda_blocks, int block_count, int blocks_per_thread ) {
	int index = (blockIdx.x * blockDim.x * blocks_per_thread) + (threadIdx.x * blocks_per_thread); // (beginning of multiprocessor segment) + (segment index).
	int i;

	// Encrypted the minimal number of blocks.
	for ( i = 0; i < blocks_per_thread; i++ ) {
		serpent_cuda_encrypt_block(&(cuda_blocks[index + i]), cuda_subkey);
	}

	// Encrypt the extra blocks that fall outside the minimal number of block.s
	index = (gridDim.x * blockDim.x * blocks_per_thread) + ((blockIdx.x * blockDim.x) + threadIdx.x); // (end of array) + (absolute thread #).
	if ( index < block_count ) {
		serpent_cuda_encrypt_block(&(cuda_blocks[index]), cuda_subkey);
	}
}


__device__ uint32 mirror_bytes32_cu(uint32 x) {
	uint32 out;

	// Change to Little Endian.
	out = (uint8_t) x;
       	out <<= 8; out |= (uint8_t) (x >> 8);
	out <<= 8; out |= (uint8_t) (x >> 16);
	out = (out << 8) | (uint8_t) (x >> 24);

	// Return out.
	return out;
}


extern "C"
int serpent_cuda_decrypt_cu(uint32* subkey, block128* blocks, int block_count) {
	// Total number of registers taken up by a single CUDA thread.
	const int REGISTERS_PER_THREAD = 8;
	block128* cuda_blocks;
	hipError_t cuda_error;
	size_t total_global_memory;
	size_t free_global_memory;
	int count; 
	int device_number;
	int multiprocessor_count;
	int thread_count;
	int i;

	// Validate parameters.
	if ( subkey == NULL ) {
		fprintf(stderr, "subkey was NULL.\n");
		return -1;
	}
	else if ( blocks == NULL ) {
		fprintf(stderr, "blocks was NULL.\n");
		return -1;
	}
	else if ( block_count < 1 ) {
		fprintf(stderr, "block_count was less than 1.\n");
		return -1;
	}

	// Get the number of devices.
	cuda_error = hipGetDeviceCount( &count );
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get device count: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	else if ( count == 0 ) {
		fprintf(stderr, "No CUDA-capable devices found.\n");
		return -1;
	}

	// Calculate multiprocessor and thread count.
	device_number = 0;
	if ( cuda_get_block_and_thread_count_max(device_number, REGISTERS_PER_THREAD, &multiprocessor_count, &thread_count) == -1 ) {
		fprintf(stderr, "Unable to get multiprocessor and thread count.\n");
		return -1;
	}

	// Move subkey to constant memory.
	cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "cuda_subkey"), subkey, sizeof(uint32) * SUBKEY_LENGTH);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to copy subkey to constant memory: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Calculate the amount of global memory available for blocks.
	cuda_error = hipMemGetInfo(&free_global_memory, &total_global_memory);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get memory information: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	free_global_memory -= SERPENT_CUDA_MEMORY_BUFFER; // Magic number.
	fprintf(stderr, "Total global memory: %i.\n", total_global_memory);

	// Calculate number of blocks per thread.
	int blocks_per_kernel = free_global_memory / sizeof(block128);
	int blocks_per_thread = blocks_per_kernel / multiprocessor_count / thread_count;
	fprintf(stderr, "Blocks global memory: %i.\nBlocks per kernel: %i.\n", free_global_memory, blocks_per_kernel);

	// Allocate a buffer for the blocks on the GPU.
	cuda_error = hipMalloc( (void**)&cuda_blocks, (int)(sizeof(block128) * blocks_per_kernel) );
	if ( cuda_error != hipSuccess ) { 
		fprintf(stderr, "Unable to malloc blocks: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Decrypt the blocks.
	i = 0;
	while (i < block_count) {
		fprintf(stderr, "Running an iteration. i: %i. block_count: %i.\n", i, block_count);

		// Corner case.
		if ( i + blocks_per_kernel > block_count ) {
			blocks_per_kernel = block_count - i;
		}

		// Move blocks to global memory.
		cuda_error = hipMemcpy( cuda_blocks, &(blocks[i]), sizeof(block128) * blocks_per_kernel, hipMemcpyHostToDevice );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to memcopy blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}

		// Run encryption.
		serpent_cuda_decrypt_blocks<<<multiprocessor_count ,thread_count>>>(cuda_blocks, blocks_per_kernel, blocks_per_thread);
		cuda_error = hipGetLastError();
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to invoke CUDA kernel: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}

		// Get blocks from global memory.
		cuda_error = hipMemcpy( &(blocks[i]), cuda_blocks, sizeof(block128) * blocks_per_kernel, hipMemcpyDeviceToHost );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to retrieve blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}
	
		// Increment i by the number of blocks processed.
		i += blocks_per_kernel;
	}

	// Free blocks from global memory.
	hipFree(cuda_blocks);

	// Return success.
	return 0;
}


extern "C"
int serpent_cuda_encrypt_cu(uint32* subkey, block128* blocks, int block_count) {
	// Maximum total number of registers taken up by a single CUDA thread.
	// This variable will need to be manually calculated and updated if
	// the algorithm implementation changes (but if you know of a way
	// to proceedurally do this, please, feel free...).
	const int REGISTERS_PER_THREAD = 8;
	//hipDeviceProp_t cuda_device;
	block128* cuda_blocks;
	hipError_t cuda_error;
	size_t total_global_memory;
	size_t free_global_memory;
	int count;
	int device_number;
	int multiprocessor_count;
	int thread_count;
	int i;

	// Validate parameters.
	if ( subkey == NULL ) {
		fprintf(stderr, "subkey was NULL.\n");
		return -1;
	}
	else if ( blocks == NULL ) {
		fprintf(stderr, "blocks was NULL.\n");
		return -1;
	}
	else if ( block_count < 1 ) {
		fprintf(stderr, "block_count was less than 1.\n");
		return -1;
	}

	// Get the number of devices.
	cuda_error = hipGetDeviceCount( &count );
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get device count: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	else if ( count == 0 ) {
		fprintf(stderr, "No CUDA-capable devices found.\n");
		return -1;
	}

	// Calculate the number of multiprocessors and threads to launch.
	device_number = 0;
	if ( cuda_get_block_and_thread_count_max(device_number, REGISTERS_PER_THREAD, &multiprocessor_count, &thread_count) == -1 ) {
		fprintf(stderr, "Unable to get max thread count.\n");
		return -1;
	}
	fprintf(stdout, "Multiprocessors: %i, threads: %i.\n", multiprocessor_count, thread_count);

	// Move subkey to constant memory.
	cuda_error = hipMemcpyToSymbol(HIP_SYMBOL( "cuda_subkey"), subkey, sizeof(uint32) * SUBKEY_LENGTH);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to copy subkey to constant memory: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Calculate the amount of global memory available for blocks.
	cuda_error = hipMemGetInfo(&free_global_memory, &total_global_memory);
	if ( cuda_error != hipSuccess ) {
		fprintf(stderr, "Unable to get memory information: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}
	free_global_memory -= SERPENT_CUDA_MEMORY_BUFFER; // Magic number.
	fprintf(stderr, "Total global memory: %i.\n", total_global_memory);

	// Calculate number of blocks per thread.
	int blocks_per_kernel = free_global_memory / sizeof(block128);
	int blocks_per_thread = (blocks_per_kernel / multiprocessor_count) / thread_count;
	fprintf(stderr, "Free global memory: %i.\nBlocks per kernel: %i.\n", free_global_memory, blocks_per_kernel);
	fprintf(stderr, "Blocks per thread: %i.\n", blocks_per_thread);

	// Allocate a buffer for the blocks on the GPU.
	if ( hipMalloc( (void**)&cuda_blocks, (int)(sizeof(block128) * blocks_per_kernel) ) != hipSuccess ) {
		fprintf(stderr, "Unable to malloc blocks: %s.\n", hipGetErrorString(cuda_error));
		return -1;
	}

	// Encrypt the blocks.
	i = 0;
	while (i < block_count) {
		fprintf(stderr, "Running an iteration. i: %i. block_count: %i.\n", i, block_count);
		// Corner case.
		if ( i + blocks_per_kernel > block_count ) {
			blocks_per_kernel = block_count - i;
			blocks_per_thread = blocks_per_kernel / multiprocessor_count / thread_count;
		}

		// Move blocks to global memory.
		cuda_error = hipMemcpy( cuda_blocks, &(blocks[i]), sizeof(block128) * blocks_per_kernel, hipMemcpyHostToDevice );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to memcopy blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}

		// Run encryption.
		serpent_cuda_encrypt_blocks<<<multiprocessor_count, thread_count>>>(cuda_blocks, blocks_per_kernel, blocks_per_thread);
		cuda_error = hipGetLastError();
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "ERROR invoking the kernel: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}

		// Get blocks from global memory.
		cuda_error = hipMemcpy( &(blocks[i]), cuda_blocks, sizeof(block128) * blocks_per_kernel, hipMemcpyDeviceToHost );
		if ( cuda_error != hipSuccess ) {
			fprintf(stderr, "Unable to retrieve blocks: %s.\n", hipGetErrorString(cuda_error));
			return -1;
		}
	
		// Increment i by the number of blocks processed.
		i += blocks_per_kernel;
	}

	// Free blocks from global memory.
	hipFree(cuda_blocks);

	// Return success.
	return 0;
}
